#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <hipfft/hipfft.h>
#include <>
#include <hip/device_functions.h>

#include "Hologram/HologramGenerator_GPU.h"
#include "graphics/sys.h"

#include <thrust/complex.h>
#include <hiprand/hiprand_kernel.h>

//hipEvent_t start, stop;

__device__ float clamp(float v, float a, float b)
{
	return v < a ? a : (v > b ? b : v);
}

__global__ void grey_normalize(float* src, uchar* dst, float* min_v, float* max_v,int nx, int ny)
{
	int tid = threadIdx.x + blockIdx.x*blockDim.x;
	int wh = nx * ny;
	if (tid < wh) {
		dst[tid] = unsigned char(clamp((src[tid] - min_v[0]) / (max_v[0] - min_v[0] + 0.000001)*255.0, 0.0, 255.0));
	}
}

__device__  void exponent_complex_gen(hipComplex* val)
{
	float exp_val = exp(val->x);
	float cos_v;
	float sin_v;
	sincos(val->y, &sin_v, &cos_v);

	val->x = exp_val * cos_v;
	val->y = exp_val * sin_v;

}

__global__ void cropFringe(int nx, int ny, hipfftComplex* in_filed, hipfftComplex* out_filed, int cropx1, int cropx2, int cropy1, int cropy2)
{
	int tid = threadIdx.x + blockIdx.x*blockDim.x;

	if (tid < nx*ny)
	{
		int x = tid % nx;
		int y = tid / nx;

		if (x >= cropx1 && x <= cropx2 && y >= cropy1 && y <= cropy2)
			out_filed[tid] = in_filed[tid];
	}
}

__global__ void kernel_fftShift(int N, int nx, int ny, hipfftComplex* input, hipfftComplex* output, bool bNormailzed)
{
	int tid = threadIdx.x + blockIdx.x*blockDim.x;

	double normalF = 1.0;
	if (bNormailzed == true)
		normalF = nx * ny;

	while (tid < N)
	{
		int i = tid % nx;
		int j = tid / nx;

		int ti = i - nx / 2; if (ti < 0) ti += nx;
		int tj = j - ny / 2; if (tj < 0) tj += ny;

		int oindex = tj * nx + ti;


		output[tid].x = input[oindex].x / normalF;
		output[tid].y = input[oindex].y / normalF;

		tid += blockDim.x * gridDim.x;
	}
}

__global__ void getFringe(int nx, int ny, hipfftComplex* in_filed, hipfftComplex* out_filed, int sig_locationx, int sig_locationy,
	double ssx, double ssy, double ppx, double ppy, double pi)
{
	int tid = threadIdx.x + blockIdx.x*blockDim.x;

	if (tid < nx*ny)
	{
		hipfftComplex shift_phase = make_hipComplex(1, 0);

		if (sig_locationy != 0)
		{
			int r = tid / nx;
			double yy = (ssy / 2.0) - (ppy)*(double)r - ppy;

			hipfftComplex val = make_hipComplex(0, 0);
			if (sig_locationy == 1)
				val.y = 2.0 * pi * (yy / (4.0 * ppy));
			else
				val.y = 2.0 * pi * (-yy / (4.0 * ppy));

			exponent_complex_gen(&val);

			shift_phase = hipCmulf(shift_phase, val);
		}

		if (sig_locationx != 0)
		{
			int c = tid % nx;
			double xx = (-ssx / 2.0) - (ppx)*(double)c - ppx;

			hipfftComplex val = make_hipComplex(0, 0);
			if (sig_locationx == -1)
				val.y = 2.0 * pi * (-xx / (4.0 * ppx));
			else
				val.y = 2.0 * pi * (xx / (4.0 * ppx));

			exponent_complex_gen(&val);
			shift_phase = hipCmulf(shift_phase, val);
		}

		out_filed[tid] = hipCmulf(in_filed[tid], shift_phase);
	}

}

__global__ void getRealPart(int nx, int ny, hipfftComplex* in_filed, float* out_filed)
{
	int tid = threadIdx.x + blockIdx.x*blockDim.x;

	if (tid < nx*ny)
		out_filed[tid] = (float)in_filed[tid].x;
}

__global__ void kernel_Merge(int nx, int ny, int num_gpu, float* src_real, float* src_imag)
{

	int tid = threadIdx.x + blockIdx.x*blockDim.x;

	if (tid >= nx * ny)	return;

	for (int i = 1; i < num_gpu; i++)
	{
		float* ptr_real = src_real + (nx*ny * 3 * i);
		float* ptr_img = src_imag + (nx*ny * 3 * i);

		atomicAdd(&src_real[tid * 3], ptr_real[tid * 3]);
		atomicAdd(&src_real[tid * 3 + 1], ptr_real[tid * 3 + 1]);
		atomicAdd(&src_real[tid * 3 + 2], ptr_real[tid * 3 + 2]);

		atomicAdd(&src_imag[tid * 3], ptr_img[tid * 3]);
		atomicAdd(&src_imag[tid * 3 + 1], ptr_img[tid * 3 + 1]);
		atomicAdd(&src_imag[tid * 3 + 2], ptr_img[tid * 3 + 2]);

	}


}

__global__ void kernel_CopytoComplex(int nx, int ny, float* src_real, float* src_img, hipfftComplex* dstR, hipfftComplex* dstG, hipfftComplex* dstB)
{
	int tid = threadIdx.x + blockIdx.x*blockDim.x;

	if (tid >= nx * ny)	return;

	dstR[tid].x = src_real[tid * 3];
	dstR[tid].y = src_img[tid * 3];

	dstG[tid].x = src_real[tid * 3+1];
	dstG[tid].y = src_img[tid * 3+1];

	dstB[tid].x = src_real[tid * 3+2];
	dstB[tid].y = src_img[tid * 3+2];

}




//__global__ void set_subhologram_position(int num_of_point, short* dev_mem, float* position_data, float len_d,
//	int nx, int ny, float pixelSize, float hcx, float hcy, float hcz)
//{
//	int tid = threadIdx.x + blockIdx.x*blockDim.x;
//	if (tid < num_of_point) {
//		float start_x_d = position_data[tid * 3] - len_d;
//		float start_y_d = position_data[tid * 3 + 1] - len_d;
//
//		dev_mem[tid * 2 + 0] = ceil((1 / pixelSize)*(start_x_d - hcx + 0.5*nx*pixelSize));
//		dev_mem[tid * 2 + 1] = ceil((1 / pixelSize)*(start_y_d - hcy + 0.5*ny*pixelSize));
//	}
//}

__global__ void set_subhologram_position(int num_of_point, short* dev_mem, float* position_data, float len_d,
	int nx, int ny, float pixelSize, float hcx, float hcy, float hcz)
{
	int tid = threadIdx.x + blockIdx.x*blockDim.x;
	if (tid < num_of_point) {
		float start_x_d = position_data[tid * 3] - len_d;
		float start_y_d = position_data[tid * 3 + 1] - len_d;

		dev_mem[tid * 2 + 0] = ceil((1 / pixelSize)*(/*start_x_d*/-len_d - hcx + 0.5*nx*pixelSize));
		dev_mem[tid * 2 + 1] = ceil((1 / pixelSize)*(/*start_y_d*/-len_d - hcy + 0.5*ny*pixelSize));
	}
}

//__global__ void point_sources_kernel_dp(int len, short*  start_pos, float* real_part_hologram, float* imagery_part_hologram,
//	float* pos, float* o_intensity, int nx_s, int ny_s, float pixelSize, float TWO_PI_DP, float waveLength, int pi)
//{
//	int tid = threadIdx.x + blockIdx.x*blockDim.x;
//	int which_hologram = blockIdx.y;
//
//	if (tid >= len * len) return;
//	//if (tid >= nx_s * ny_s) return;
//
//	int pi_index = pi + which_hologram;
//	int i, j;
//	i = start_pos[pi_index * 2] + tid % len;
//	j = start_pos[pi_index * 2 + 1] + tid / len;
//	//i = tid % nx_s;
//	//j = tid / nx_s;
//
//	if (i < 0 || i >= nx_s || j < 0 || j >= ny_s) return;
//
//	float x, y, z;
//	//x = (i - 1) * pixelSize - 0.5*nx_s*pixelSize;
//	//y = (ny_s - j) * pixelSize - 0.5*ny_s*pixelSize;
//
//	float x0 = 0, y0 = 0, z0 = 0;
//	x = i * pixelSize - 0.5*nx_s*pixelSize + x0;
//	y = j * pixelSize - 0.5*ny_s*pixelSize + y0;
//	z = z0;
//
//	float pcx = pos[pi_index * 3];
//	float pcy = pos[pi_index * 3 + 1];
//	float pcz = pos[pi_index * 3 + 2];
//
//	//=== check if the position z is a effective point ===================================
//	/*
//	float len_ccp, len_cp;
//
//	float n = sqrt((x - pcx)*(x - pcx) + (y - pcy)*(y - pcy));
//	float unitx = (x - pcx) / n;
//	float unity = (y - pcy) / n;
//	float ccx = x + pixelSize * unitx;
//	float ccy = y + pixelSize * unity;
//
//	float l = sqrt((x - pcx)*(x - pcx) + (y - pcy)*(y - pcy));
//	len_cp = sqrt(pcz*pcz + l * l);
//
//	float ll = sqrt((ccx - pcx)*(ccx - pcx) + (ccy - pcy)*(ccy - pcy));
//	len_ccp = sqrt(pcz*pcz + ll * ll);
//
//	if (abs(len_ccp - len_cp) > waveLength) {
//		return;
//	}*/
//
//	//=================================================================================
//	
//	float3 val;
//	val.x = o_intensity[pi_index * 3 + 0] / waveLength;
//	val.y = o_intensity[pi_index * 3 + 1] / waveLength;
//	val.z = o_intensity[pi_index * 3 + 2] / waveLength;
//
//	float oz = (int)(pcz / waveLength) * (waveLength);
//	float rx = x - pcx;
//	float ry = y - pcy;
//	float rz = z - oz;
//	const float l2 = sqrtf(rx*rx + ry*ry + rz*rz);
//
//	float sval = (float)(TWO_PI_DP / waveLength * l2);
//	float cos_v = cos(sval);
//	float sin_v = sin(sval);
//
//	int index = (j * nx_s * 3 + i * 3);
//
//	atomicAdd(&real_part_hologram[index], (val.x * cos_v));
//	atomicAdd(&real_part_hologram[index + 1], (val.y * cos_v));
//	atomicAdd(&real_part_hologram[index + 2], (val.z * cos_v));
//
//	atomicAdd(&imagery_part_hologram[index], (-val.x * sin_v));
//	atomicAdd(&imagery_part_hologram[index + 1], (-val.y * sin_v));
//	atomicAdd(&imagery_part_hologram[index + 2], (-val.z * sin_v));
//	
//}

__global__ void point_sources_kernel_dp(int len, short*  start_pos, float* real_part_hologram, float* imagery_part_hologram,
	float* pos, float* o_intensity, int nx_s, int ny_s, float pixelSize, float TWO_PI_DP, /*double* waveLength,*/ int pi)//JS
{
	int tid = threadIdx.x + blockIdx.x*blockDim.x;
	int which_hologram = blockIdx.y;

	if (tid >= len * len) return;
	//if (tid >= nx_s * ny_s) return;

	int pi_index = pi + which_hologram;
	int i, j;//i,j�� �ȼ��ε�������
	i = start_pos[pi_index * 2] + tid % len;
	j = start_pos[pi_index * 2 + 1] + tid / len;
	//i = tid % nx_s;
	//j = tid / nx_s;

	if (i < 0 || i >= nx_s || j < 0 || j >= ny_s) return;

	//float x0 = pixelSize / 2, y0 = pixelSize / 2, z0 = 0;
	float xx = 0, yy = 0;// , z0 = 0;
	xx = i * pixelSize - 0.5*nx_s*pixelSize;
	yy = j * pixelSize - 0.5*ny_s*pixelSize;//�ø�?��������
	//zx = z0;
	float waveLength[3] = { 660 * 1e-9 ,532 * 1e-9 ,457 * 1e-9 };

	float x = pos[pi_index * 3];
	float y = pos[pi_index * 3 + 1];
	float z = pos[pi_index * 3 + 2];

	float k;

	float p_A;
	float p_phi=0;// = 2 * pi / waveLength[0] * z;
	float p_xyz;
	//float r = sqrt(pow(x - xx, 2) + pow(y - yy, 2) + pow(z, 2));

	int index = ((nx_s - j - 1) * nx_s * 3 + i * 3);//UDflip

	for (int i = 0; i < 3; i++)
	{

		if (abs(x - xx) / (waveLength[i] * z) < (float)1 / ((float)2 * pixelSize) && abs(y - yy) / (waveLength[i] * z) < (float)1 / ((float)4 * pixelSize))//�� ������ 2 ������ 4��
		{
			k = TWO_PI_DP / waveLength[i];
			//p_phi = k * r;
			p_A= o_intensity[pi_index * 3 + i]*cos(k*((pow(x - xx, 2) + pow(y - yy, 2)) / ((float)2 * z) + waveLength[i] * yy / ((float)4 * pixelSize)) + p_phi);
			atomicAdd(&real_part_hologram[index + i], p_A);
			//atomicAdd(&imagery_part_hologram[index + i], (CGH.imag()));
		}
	}
	/*
	float xa, ya, za;
	//x = (i - 1) * pixelSize - 0.5*nx_s*pixelSize;-
	//y = (ny_s - j) * pixelSize - 0.5*ny_s*pixelSize;

	float x0 = pixelSize / 2, y0 = pixelSize / 2, z0 = 0;
	//float x0 = 0, y0 = 0, z0 = 0;
	xa = i * pixelSize - 0.5*nx_s*pixelSize + x0;
	ya = j * pixelSize - 0.5*ny_s*pixelSize + y0;
	za = z0;

	float x = pos[pi_index * 3];
	float y = pos[pi_index * 3 + 1];
	float z = pos[pi_index * 3 + 2];

	//float nm = 1e-9;
	//float um = 1e-6;
	//float mm = 1e-3;
	//float cm = 1e-2;

	//norm_v = [0 0 - 1];

	float djh = 0 * 1e-2;

	float dx = pixelSize; //�ϴ� �ӽ÷� x�ȼ���ġ�� �Ѵ�����
	float dy = pixelSize;

	float X_Shift = 0;
	//float Y_Shift = 0.5;
	float Y_Shift = -0.5;

	//thrust::complex<float> tmp(0, 0);
	thrust::complex<float> imgI(0, 1);
	//hipDoubleComplex imgI2 = make_hipDoubleComplex(0, 2);
	hiprandState cs;
	hiprand_init(pi_index, 0, 0, &cs);
	float rand = hiprand_uniform(&cs);
	float B = -1;
	float R;
	thrust::complex<float> i2PI = imgI * TWO_PI_DP;
	thrust::complex<float> A;
	thrust::complex<float> CGH;
	//rand = dis(gen);
	//int index = (j * nx_s * 3 + i * 3);
	int index = ((nx_s - j - 1) * nx_s * 3 + i * 3);//UDflip
	//int index = (j * nx_s * 3 + (nx_s-i-1) * 3);//LRflip

	//float Rwl = 660 * nm;
	//float Gwl = 532 * nm;
	//float Bwl = 457 * nm;

	//float lambda = Rwl; //wavelength������
	float waveLength[3] = { 660 * 1e-9 ,532 * 1e-9 ,457 * 1e-9 };

	float theta_x = asin((waveLength[2] / (2 * dx))); // diffraction angle
	float theta_y = asin((waveLength[2] / (2 * dy)));

	float drx = abs(djh - z)*tan(theta_x);
	float dry = abs(djh - z)*tan(theta_y);

	if (pow((xa - x) / drx, 2) + pow((ya - y) / dry, 2) <= 1)
	{
		R = (-z / abs(z))*sqrt(pow(djh - z, 2) + pow(xa - x, 2) + pow(ya - y, 2));
		for (int i = 0; i < 3; i++)
		{
			A = exp(i2PI*rand)*exp(i2PI*R / waveLength[i]) / R * o_intensity[pi_index * 3 + i];//intensity�� A�� ��������� �����ش� ���򺰷�
			CGH = A * B*(dx*dy) / (imgI*waveLength[i])*exp(i2PI*(xa / (2 * dx)*X_Shift*(waveLength[2] / waveLength[i]) + ya / (2 * dy)*Y_Shift*(waveLength[2] / waveLength[i])));
			atomicAdd(&real_part_hologram[index + i], (CGH.real()));
			atomicAdd(&imagery_part_hologram[index + i], (CGH.imag()));
		}
	}*/
}

//__global__ void point_sources_kernel_dp(int len, short*  start_pos, float* real_part_hologram, float* imagery_part_hologram,
//	float* pos, float* o_intensity, int nx_s, int ny_s, float pixelSize, float TWO_PI_DP, /*double* waveLength,*/ int pi)//JH
//{
//	int tid = threadIdx.x + blockIdx.x*blockDim.x;
//	int which_hologram = blockIdx.y;
//
//	if (tid >= len * len) return;
//	//if (tid >= nx_s * ny_s) return;
//
//	int pi_index = pi + which_hologram;
//	int i, j;//i,j�� �ȼ��ε�������
//	i = start_pos[pi_index * 2] + tid % len;
//	j = start_pos[pi_index * 2 + 1] + tid / len;
//	//i = tid % nx_s;
//	//j = tid / nx_s;
//
//	if (i < 0 || i >= nx_s || j < 0 || j >= ny_s) return;
//
//	float xa, ya, za;
//	//x = (i - 1) * pixelSize - 0.5*nx_s*pixelSize;-
//	//y = (ny_s - j) * pixelSize - 0.5*ny_s*pixelSize;
//
//	float x0 = pixelSize / 2, y0 = pixelSize / 2, z0 = 0;
//	//float x0 = 0, y0 = 0, z0 = 0;
//	xa = i * pixelSize - 0.5*nx_s*pixelSize + x0;
//	ya = j * pixelSize - 0.5*ny_s*pixelSize + y0;
//	za = z0;
//
//	float x = pos[pi_index * 3];
//	float y = pos[pi_index * 3 + 1];
//	float z = pos[pi_index * 3 + 2];
//
//	//float nm = 1e-9;
//	//float um = 1e-6;
//	//float mm = 1e-3;
//	//float cm = 1e-2;
//
//	//norm_v = [0 0 - 1];
//
//	float djh = 0 * 1e-2;
//
//	float dx = pixelSize; //�ϴ� �ӽ÷� x�ȼ���ġ�� �Ѵ�����
//	float dy = pixelSize;
//
//	float X_Shift = 0;
//	//float Y_Shift = 0.5;
//	float Y_Shift = -0.5;
//
//	//thrust::complex<float> tmp(0, 0);
//	thrust::complex<float> imgI(0, 1);
//	//hipDoubleComplex imgI2 = make_hipDoubleComplex(0, 2);
//	hiprandState cs;
//	hiprand_init(pi_index, 0, 0, &cs);
//	float rand = hiprand_uniform(&cs);
//	float B = -1;
//	float R;
//	thrust::complex<float> i2PI = imgI * TWO_PI_DP;
//	thrust::complex<float> A;
//	thrust::complex<float> CGH;
//	//rand = dis(gen);
//	//int index = (j * nx_s * 3 + i * 3);
//	int index = ((nx_s - j - 1) * nx_s * 3 + i * 3);//UDflip
//	//int index = (j * nx_s * 3 + (nx_s-i-1) * 3);//LRflip
//
//	//float Rwl = 660 * nm;
//	//float Gwl = 532 * nm;
//	//float Bwl = 457 * nm;
//
//	//float lambda = Rwl; //wavelength������
//	float waveLength[3] = { 660 * 1e-9 ,532 * 1e-9 ,457 * 1e-9 };
//
//	float theta_x = asin((waveLength[2] / (2 * dx))); // diffraction angle
//	float theta_y = asin((waveLength[2] / (2 * dy)));
//
//	float drx = abs(djh - z)*tan(theta_x);
//	float dry = abs(djh - z)*tan(theta_y);
//
//	//thrust::complex<float> tempjj;
//	//if (pow((xa - x) / drx, 2) + pow((ya - y) / dry, 2) <= 1)
//	//{
//	//	R = (-z / abs(z))*sqrt(pow(djh - z, 2) + pow(xa - x, 2) + pow(ya - y, 2));
//	//	A = exp(i2PI*rand) / R;
//	//	CGH = B * (dx*dy) / imgI;
//	//	tempjj = i2PI * (xa / (2 * dx)*X_Shift + ya / (2 * dy)*Y_Shift);
//	//	for (int i = 0; i < 3; i++)
//	//	{
//	//		A = A*exp(i2PI*R / waveLength[i]) * o_intensity[pi_index * 3 + i];//intensity�� A�� ��������� �����ش� ���򺰷�
//	//		CGH = A *CGH / waveLength[i]*exp(tempjj*(waveLength[2] / waveLength[i]));
//	//		atomicAdd(&real_part_hologram[index + i], (CGH.real()));
//	//		atomicAdd(&imagery_part_hologram[index + i], (CGH.imag()));
//	//	}
//	//}
//	
//	if (pow((xa - x) / drx, 2) + pow((ya - y) / dry, 2) <= 1)
//	{
//		R = (-z / abs(z))*sqrt(pow(djh - z, 2) + pow(xa - x, 2) + pow(ya - y, 2));
//		for (int i = 0; i < 3; i++)
//		{
//			A = exp(i2PI*rand)*exp(i2PI*R / waveLength[i]) / R * o_intensity[pi_index * 3 + i];//intensity�� A�� ��������� �����ش� ���򺰷�
//			CGH = A * B*(dx*dy) / (imgI*waveLength[i])*exp(i2PI*(xa / (2 * dx)*X_Shift*(waveLength[2] / waveLength[i]) + ya / (2 * dy)*Y_Shift*(waveLength[2] / waveLength[i])));
//			atomicAdd(&real_part_hologram[index + i], (CGH.real()));
//			atomicAdd(&imagery_part_hologram[index + i], (CGH.imag()));
//		}
//	}
//}


#include <QtGui/QImage>

void HologramGenerator::point_sources_method_xy_plane_CUDA()
{
	const int nx = config_PC_->pixel_number[0];
	const int ny = config_PC_->pixel_number[1];
	const int N = nx * ny;
	const int M = data_PC_->n_points;

	//if (!start)			hipEventCreate(&start);
	//if (!stop)			hipEventCreate(&stop);

	for (int gpu_i = 0; gpu_i < num_gpu_; gpu_i++)
	{
		SetCurrentGPU(gpu_i);

		HANDLE_ERROR(hipMemsetAsync(save_a_d_[gpu_i], 0, sizeof(float)*N* 3));
		HANDLE_ERROR(hipMemsetAsync(save_b_d_[gpu_i], 0, sizeof(float)*N* 3));

		HANDLE_ERROR(hipMemcpyAsync(obj_position_d_[gpu_i], data_PC_->ObjPosition, sizeof(float)* M * 3, hipMemcpyHostToDevice));
		HANDLE_ERROR(hipMemcpyAsync(obj_intensity_d_[gpu_i], data_PC_->ObjIntensity, sizeof(float)* M * 3, hipMemcpyHostToDevice));

	}

	const double pixelSize = config_PC_->pixel_pitch[0];
	const double hcx = 0.0;
	const double hcy = 0.0;
	const double hcz = 0.0;
	const double waveLength = config_PC_->wave_length;

	unsigned int nblocks;

#ifdef DEBUGMSG
	LOG("object number %d\n", M);
#endif

	int len = 0;
	double len_d;
	double pz = data_PC_->ObjPosition[2];

	//find_size(len_d, len, pz, pixelSize, hcz, waveLength);
	find_size(len_d, len, pz, pixelSize, hcz);//JH
	//LOG("Sub hologram size=%d # of points=%d\n", len, M);

	nblocks = (M + kBlockThreads - 1) / kBlockThreads;

	for (int gpu_i = 0; gpu_i < num_gpu_; gpu_i++)
	{
		SetCurrentGPU(gpu_i);
		set_subhologram_position << <nblocks, kBlockThreads, 0 >> >
			(M, obj_hologram_pos_[gpu_i], obj_position_d_[gpu_i], len_d, nx, ny, pixelSize, hcx, hcy, hcz);

	}

	int* block_threads = new int[num_gpu_];
	int* work_load = new int[num_gpu_];

	for (int gpu_i = 0; gpu_i < num_gpu_; gpu_i++)
	{
		SetCurrentGPU(gpu_i);

		block_threads[gpu_i] = 256;
		work_load[gpu_i] = int(float(len * len) / float(core_cnt_[gpu_i]) + 0.5) * core_cnt_[gpu_i];
		while (core_cnt_[gpu_i] % block_threads[gpu_i] != 0) {
			block_threads[gpu_i]--;
		}
#ifdef DEBUGMSG
		_cprintf("sub holo size %d, work_load %d, block threads %d\n", len, work_load[gpu_i], block_threads[gpu_i]);
#endif
	}

	int pi = 0;

	while (pi < M - (kPointBundle*num_gpu_))
	{
		for (int gpu_i = 0; gpu_i < num_gpu_ ; gpu_i++)
		{
			SetCurrentGPU(gpu_i);

			dim3 grid((work_load[gpu_i] + block_threads[gpu_i] - 1) / block_threads[gpu_i], kPointBundle, 1);

			//LOG("M:%d, pi:%d, grid:%d %d %d\n", M, pi, grid.x, grid.y, grid.z);

			/*point_sources_kernel_dp << < grid, block_threads[gpu_i], 0 >> >
				(len, obj_hologram_pos_[gpu_i], save_a_d_[gpu_i], save_b_d_[gpu_i], obj_position_d_[gpu_i], obj_intensity_d_[gpu_i],
					nx, ny, pixelSize, TWO_PI_DP, waveLength, pi);*/

			point_sources_kernel_dp << < grid, block_threads[gpu_i], 0 >> >
				(len, obj_hologram_pos_[gpu_i], save_a_d_[gpu_i], save_b_d_[gpu_i], obj_position_d_[gpu_i], obj_intensity_d_[gpu_i],
					nx, ny, pixelSize, TWO_PI_DP, pi);//BJJ

			pi += kPointBundle;
		}
	}
	
	if (pi < M) {

		for (int gpu_i = 0; gpu_i < num_gpu_ && pi < M; gpu_i++)
		{
			SetCurrentGPU(gpu_i);

			dim3 grid((work_load[gpu_i] + block_threads[gpu_i] - 1) / block_threads[gpu_i], kPointBundle, 1);
			if (pi + kPointBundle >= M)
				grid.y = M - pi;

			//LOG("M:%d, pi:%d, grid:%d %d %d\n", M, pi, grid.x, grid.y, grid.z);

			/*point_sources_kernel_dp << < grid, block_threads[gpu_i], 0 >> >
				(len, obj_hologram_pos_[gpu_i], save_a_d_[gpu_i], save_b_d_[gpu_i], obj_position_d_[gpu_i], obj_intensity_d_[gpu_i],
					nx, ny, pixelSize, TWO_PI_DP, waveLength, pi);*/
			point_sources_kernel_dp << < grid, block_threads[gpu_i], 0 >> >
				(len, obj_hologram_pos_[gpu_i], save_a_d_[gpu_i], save_b_d_[gpu_i], obj_position_d_[gpu_i], obj_intensity_d_[gpu_i],
					nx, ny, pixelSize, TWO_PI_DP, pi);//BJJ

			pi += kPointBundle;
		}

	}


	hipSetDevice(0);
	HANDLE_ERROR(hipMemcpyAsync(cghFieldReal_GPU_, save_a_d_[0], N * sizeof(float) * 3, hipMemcpyDeviceToDevice));
	HANDLE_ERROR(hipMemcpyAsync(cghFieldImag_GPU_, save_b_d_[0], N * sizeof(float) * 3, hipMemcpyDeviceToDevice));

	if (num_gpu_ > 1)
	{
		for (int gpu_i = 1; gpu_i < num_gpu_; gpu_i++)
		{
			hipSetDevice(gpu_i);

			float * ptr_real = cghFieldReal_GPU_ + (N * 3 * gpu_i);
			float * ptr_imag = cghFieldImag_GPU_ + (N * 3 * gpu_i);

			HANDLE_ERROR(hipMemcpyPeerAsync(ptr_real, 0, save_a_d_[gpu_i], gpu_i, N * sizeof(float) * 3));
			HANDLE_ERROR(hipMemcpyPeerAsync(ptr_imag, 0, save_b_d_[gpu_i], gpu_i, N * sizeof(float) * 3));
		}

	}

	hipSetDevice(0);
	//HANDLE_ERROR(hipMemsetAsync(complex_H_GPU_, 0, sizeof(hipfftComplex)*N));
	HANDLE_ERROR(hipMemsetAsync(complex_H_GPU_R_, 0, sizeof(hipfftComplex)*N));
	HANDLE_ERROR(hipMemsetAsync(complex_H_GPU_G_, 0, sizeof(hipfftComplex)*N));
	HANDLE_ERROR(hipMemsetAsync(complex_H_GPU_B_, 0, sizeof(hipfftComplex)*N));

	//cudaMergeNCopytoComplex(nx, ny, num_gpu_, cghFieldReal_GPU_, cghFieldImag_GPU_, complex_H_GPU_);
	cudaMergeNCopytoComplex(nx, ny, num_gpu_, cghFieldReal_GPU_, cghFieldImag_GPU_, complex_H_GPU_R_, complex_H_GPU_G_, complex_H_GPU_B_);

	delete[] block_threads, work_load;
}






void HologramGenerator::clear_GPU_Var()
{
	for (int gpu_i = 0; gpu_i < num_gpu_; gpu_i++) 
	{
		SetCurrentGPU(gpu_i);

		if (save_a_d_[gpu_i])			hipFree(save_a_d_[gpu_i]);
		if (save_b_d_[gpu_i])			hipFree(save_b_d_[gpu_i]);
		if (obj_hologram_pos_[gpu_i])	hipFree(obj_hologram_pos_[gpu_i]);
		if (obj_position_d_[gpu_i])		hipFree(obj_position_d_[gpu_i]);
		if (obj_intensity_d_[gpu_i])	hipFree(obj_intensity_d_[gpu_i]);
		
		save_a_d_[gpu_i] = 0;
		save_b_d_[gpu_i] = 0;

		obj_position_d_[gpu_i] = 0;
		obj_intensity_d_[gpu_i] = 0;
		obj_hologram_pos_[gpu_i] = 0;

	}

	SetCurrentGPU(0);

	if (cghFieldReal_GPU_)	hipFree(cghFieldReal_GPU_);
	if (cghFieldImag_GPU_)	hipFree(cghFieldImag_GPU_);
	cghFieldReal_GPU_ = 0;
	cghFieldImag_GPU_ = 0;


	if (k_temp_d_)		HANDLE_ERROR(hipFree(k_temp_d_));
	if (k_temp2_d_)		HANDLE_ERROR(hipFree(k_temp2_d_));
	//if (complex_H_GPU_)	HANDLE_ERROR(hipFree(complex_H_GPU_));
	if (complex_H_GPU_R_)	HANDLE_ERROR(hipFree(complex_H_GPU_R_));
	if (complex_H_GPU_G_)	HANDLE_ERROR(hipFree(complex_H_GPU_G_));
	if (complex_H_GPU_B_)	HANDLE_ERROR(hipFree(complex_H_GPU_B_));
	k_temp_d_ = 0;
	k_temp2_d_ = 0;
	//complex_H_GPU_ = 0;
	complex_H_GPU_R_ = 0;
	complex_H_GPU_G_ = 0;
	complex_H_GPU_B_ = 0;

	//if (holo_encoded_GPU_)		HANDLE_ERROR(hipFree(holo_encoded_GPU_));
	if (holo_encoded_GPU_R_)		HANDLE_ERROR(hipFree(holo_encoded_GPU_R_));
	if (holo_encoded_GPU_G_)		HANDLE_ERROR(hipFree(holo_encoded_GPU_G_));
	if (holo_encoded_GPU_B_)		HANDLE_ERROR(hipFree(holo_encoded_GPU_B_));
	if (holo_normalized_GPU_)	HANDLE_ERROR(hipFree(holo_normalized_GPU_));
	//holo_encoded_GPU_ = 0;
	holo_encoded_GPU_R_ = 0;
	holo_encoded_GPU_G_ = 0;
	holo_encoded_GPU_B_ = 0;
	holo_normalized_GPU_ = 0;

	if (reduce_source_) delete reduce_source_;
	if (reduce_min_) delete reduce_min_;
	if (reduce_max_) delete reduce_max_;

	reduce_source_ = 0;
	reduce_min_ = 0;
	reduce_max_ = 0;

}



void HologramGenerator::normalize_gpu()
{
	ivec2 pn = config_PC_->pixel_number;
	int pnx = pn[0];
	int pny = pn[1];
	int N = pnx * pny;

	SetCurrentGPU(0);

	//reduce_source_->set_mem(holo_encoded_GPU_);
	float* min_v_mem;
	float* max_v_mem;
	hipMalloc(&min_v_mem, sizeof(float));
	hipMalloc(&max_v_mem, sizeof(float));
	reduce_min_->set_mem(min_v_mem);
	reduce_max_->set_mem(max_v_mem);
	reduce_source_->min_reduce(0, reduce_min_, 1.0);
	reduce_source_->max_reduce(0, reduce_max_, 1.0);

	dim3 grid((N + kBlockThreads - 1) / kBlockThreads, 1, 1);
	//grey_normalize << < grid, kBlockThreads, 0 >> > (holo_encoded_GPU_, holo_normalized_GPU_, min_v_mem, max_v_mem, pnx, pny);
	hipMemcpy(holo_normalized_, holo_normalized_GPU_, N * sizeof(uchar), hipMemcpyDeviceToHost);

	hipFree(min_v_mem);
	hipFree(max_v_mem);

}
void grey_normalize_api(float* src, uchar* dst, float* min_v, float* max_v, int nx, int ny)
{
    int N = nx * ny;
    dim3 grid((N + kBlockThreads - 1) / kBlockThreads, 1, 1);
    grey_normalize << < grid, kBlockThreads, 0 >> > (src, dst, min_v, max_v, nx, ny);

}
void HologramGenerator::cudaCropFringe(int nx, int ny, hipfftComplex* in_field, hipfftComplex* out_field, int cropx1, int cropx2, int cropy1, int cropy2)
{
	unsigned int nblocks = (nx*ny + kBlockThreads - 1) / kBlockThreads;

	cropFringe << < nblocks, kBlockThreads, 0 >> > (nx, ny, in_field, out_field, cropx1, cropx2, cropy1, cropy2);
}

void HologramGenerator::cudaFFT(int nx, int ny, hipfftComplex* in_field, hipfftComplex* output_field, int direction, bool bNormalized)
{
	unsigned int nblocks = (nx*ny + kBlockThreads - 1) / kBlockThreads;
	int N = nx * ny;
	kernel_fftShift << <nblocks, kBlockThreads, 0 >> > (N, nx, ny, in_field, output_field, false);

	hipfftHandle plan;

	// fft
	if (hipfftPlan2d(&plan, ny, nx, HIPFFT_C2C) != HIPFFT_SUCCESS)
	{
		//LOG("FAIL in creating cufft plan");
		return;
	};

	hipfftResult result;

	if (direction == -1)
		result = hipfftExecC2C(plan, output_field, in_field, HIPFFT_FORWARD);
	else
		result = hipfftExecC2C(plan, output_field, in_field, HIPFFT_BACKWARD);

	if (result != HIPFFT_SUCCESS)
	{
		//LOG("------------------FAIL: execute cufft, code=%s", result);
		return;
	}

	if (hipDeviceSynchronize() != hipSuccess) {
		//LOG("Cuda error: Failed to synchronize\n");
		return;
	}

	kernel_fftShift << < nblocks, kBlockThreads, 0 >> > (N, nx, ny, in_field, output_field, bNormalized);

	hipfftDestroy(plan);
}

void HologramGenerator::cudaGetFringe(int pnx, int pny, hipfftComplex* in_field, hipfftComplex* out_field, int sig_locationx, int sig_locationy,
	double ssx, double ssy, double ppx, double ppy, double PI)
{
	unsigned int nblocks = (pnx*pny + kBlockThreads - 1) / kBlockThreads;

	getFringe << < nblocks, kBlockThreads, 0 >> > (pnx, pny, in_field, out_field, sig_locationx, sig_locationy, ssx, ssy, ppx, ppy, PI);
}

void HologramGenerator::cudaGetRealpart(int pnx, int pny, hipfftComplex* in_field, float* out_field)
{
	unsigned int nblocks = (pnx*pny + kBlockThreads - 1) / kBlockThreads;

	getRealPart << < nblocks, kBlockThreads, 0 >> > (pnx, pny, in_field, out_field);
	   
}

void HologramGenerator::cudaMergeNCopytoComplex(int pnx, int pny, int num_gpu, float* src_real, float* src_imag, hipfftComplex* dstR, hipfftComplex* dstG, hipfftComplex* dstB)
{
	unsigned int nblocks = (pnx*pny + kBlockThreads - 1) / kBlockThreads;

	kernel_Merge << < nblocks, kBlockThreads, 0 >> > (pnx, pny, num_gpu, src_real, src_imag);


	kernel_CopytoComplex << < nblocks, kBlockThreads, 0 >> > (pnx, pny, src_real, src_imag, dstR, dstG, dstB);
}


